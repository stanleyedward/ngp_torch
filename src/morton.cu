// define morton code encoding/decoding
#include "hip/hip_vector_types.h"
#include "utils.h"
#include "pcg32.h"
#include <torch/extension.h>


torch::Tensor morton3D_cu(const torch::Tensor coords){
    return coords;
}

torch::Tensor morton3D_invert_cu(const torch::Tensor indices){
    return indices;
}